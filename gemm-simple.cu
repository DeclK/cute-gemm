#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <cute/util/gpu_clock.hpp>

template <typename T>
void gen_rand_data(T *data, int n);

template <typename T, int kTileM, int kTileN, int kTileK, typename TiledMMA>
__global__ void gemm_simple(T *Cptr, const T *Aptr, const T *Bptr, int m, int n, int k) {

  using namespace cute;

  Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
  Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
  Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(m, n), make_stride(n, Int<1>{}));

  int ix = blockIdx.x;
  int iy = blockIdx.y;

  Tensor gA = local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(iy, _));
  Tensor gB = local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(ix, _));
  Tensor gC = local_tile(C, make_tile(Int<kTileM>{}, Int<kTileN>{}), make_coord(iy, ix));
  // gA(kTileM, kTileK, num_tile_k) (128, 32, 256 / 32) = (128, 32, 8)
  // gB(kTileN, kTileK, num_tile_k) (128, 32, 256 / 32) = (128, 32, 8)
  // gC(kTileM, kTileN) (128, 128) = (128, 128)
  TiledMMA tiled_mma;
  auto thr_mma = tiled_mma.get_slice(threadIdx.x);
  // tiled mma can now deal with problem size (M, N, K) = (32, 16, 16)
  auto tAgA = thr_mma.partition_A(gA); // (MMA, MMA_M, MMA_K, num_tile_k) (8, 4, 2, 8) = ((2, 2, 2), 128 / 32, 32 / 16, 8)
  auto tBgB = thr_mma.partition_B(gB); // (MMA, MMA_N, MMA_K, num_tile_k) (4, 8, 2, 8) = ((2, 4, 2), 128 / 16, 32 / 16, 8)
  auto tCgC = thr_mma.partition_C(gC); // (MMA, MMA_M, MMA_N) (4, 4, 8) = ((2, 2, 2), 128 / 32, 128 / 16)
  // print the shape of tAgA
  // printf("tAgA shape = %d %d %d %d\n", size<0>(tAgA), size<1>(tAgA), size<2>(tAgA), size<3>(tAgA));
  #if 1
  if (thread0() && block0()) {
  print("tAgA shape = "); print(tAgA); print("\n");
  print("tBgB shape = "); print(tBgB); print("\n");
  print("tCgC shape = "); print(tCgC); print("\n");
  }
  #endif

  auto tArA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
  auto tBrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
  auto tCrC = thr_mma.partition_fragment_C(gC(_, _));     // (MMA, MMA_M, MMA_N)
 
  clear(tCrC);
  
  int num_tile_k = size<2>(gA);
#pragma unroll 1
  for(int itile = 0; itile < num_tile_k; ++itile) {
    cute::copy(tAgA(_, _, _, itile), tArA);
    cute::copy(tBgB(_, _, _, itile), tBrB);

    cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
  }

  cute::copy(tCrC, tCgC); 
}

int main() {
  srand(10086);

  using T = cute::half_t;
  using namespace cute;

  T *Cptr;
  T *Aptr;
  T *Bptr;

  // for learning purpose, I set the m, n, k the same as the problem size that
  // a single threadblock can handle
  int m = 128;
  int n = 128;
  int k = 32;

  hipMalloc(&Cptr, sizeof(T) * m * n);
  hipMalloc(&Aptr, sizeof(T) * m * k);
  hipMalloc(&Bptr, sizeof(T) * k * n);

  T *Aptr_host;
  T *Bptr_host;
  Aptr_host = (T*)malloc(sizeof(T) * m * k);
  Bptr_host = (T*)malloc(sizeof(T) * n * k);
  gen_rand_data(Aptr_host, m * k);
  gen_rand_data(Bptr_host, n * k);

  hipMemcpy(Aptr, Aptr_host, sizeof(T) * m * k, hipMemcpyHostToDevice);
  hipMemcpy(Bptr, Bptr_host, sizeof(T) * n * k, hipMemcpyHostToDevice);

  using mma_op = SM80_16x8x16_F16F16F16F16_TN;
  using mma_traits = MMA_Traits<mma_op>;
  using mma_atom = MMA_Atom<mma_traits>;

  using MMA = decltype(make_tiled_mma(mma_atom{}, 
                      make_layout(Shape<_1, _1, _1>{}), 
                      make_layout(Shape<_1, _1, _1>{})));
  // MMA mma_tmp;
  // print_latex(mma_tmp);
  constexpr int kTileM = 128; 
  constexpr int kTileN = 128; 
  constexpr int kTileK = 32; 

  dim3 block(size(MMA{}));
  dim3 grid(n / kTileN, m / kTileM);
  
  // cute gemm
  GPU_Clock timer;
  int num_iter = 1;
  timer.start();

  for (int i = 0; i < num_iter; ++i) {
    gemm_simple<T, kTileM, kTileN, kTileK, MMA><<<grid, block>>>(Cptr, Aptr, Bptr, m, n, k);
  }
  hipDeviceSynchronize();

  double elapsed_time_ms = timer.seconds() / num_iter * 1000;
  printf("elapsed time cute = %f ms\n", elapsed_time_ms);
  auto err = hipGetLastError();
  if (err) printf("err = %d, str = %s\n", err, hipGetErrorString(err));

  // cublas
  T *Cptr_cublas;

  hipMalloc(&Cptr_cublas, sizeof(T) * m * n);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  half alpha = half(1.f);
  half beta = half(0.f);
  timer.start();
  for (int i = 0; i < num_iter; ++i) {
    hipblasStatus_t ret = hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
          	  n, m, k,
          	  &alpha,
          	  (half *)Bptr, k,
          	  (half *)Aptr, k,
          	  &beta,
          	  (half *)Cptr_cublas, n);
    if (ret != HIPBLAS_STATUS_SUCCESS) {
      printf("blas err = %d, str = %s\n", ret, cublasGetStatusString(ret));
    }
  }

  hipDeviceSynchronize();
  elapsed_time_ms = timer.seconds() / num_iter * 1000;
  printf("elapsed time cublas = %f ms\n", elapsed_time_ms);
  err = hipGetLastError();
  if (err) printf("err = %d, str = %s\n", err, hipGetErrorString(err));

  T *Cptr_host;
  T *Cptr_cublas_host;

  Cptr_host = (T*)malloc(sizeof(T) * m * n);
  Cptr_cublas_host = (T*)malloc(sizeof(T) * m * n);

  // compare
  hipMemcpy(Cptr_host, Cptr, sizeof(T) * m * n, hipMemcpyDeviceToHost);
  hipMemcpy(Cptr_cublas_host, Cptr_cublas, sizeof(T) * m * n, hipMemcpyDeviceToHost);

  float threshold = 0.1;
  for (int i = 0; i < m * n; ++i) {
    float v1 = Cptr_host[i];
    float v2 = Cptr_cublas_host[i];
    if (fabs(v2 - v1) > threshold) {
      printf("NOT EQUAL: v1 = %f, v2 = %f\n", v1, v2);
    }
  }

  Tensor tensor_C = make_tensor(Cptr_host, make_shape(m, n), make_stride(n, 1));
  Tensor tensor_C_cublas = make_tensor(Cptr_cublas_host, make_shape(m, n), make_stride(n, 1));

  auto tile = make_tile(8, 8);
  auto coor = make_coord(0, 0);
  Tensor tc1 = local_tile(tensor_C, tile, coor);
  Tensor tc1_cublas = local_tile(tensor_C_cublas, tile, coor);

  // print_tensor(tc1);
  // print_tensor(tc1_cublas);
}

template <typename T>
void gen_rand_data(T *data, int n) {
  for (int i = 0; i < n; ++i) {
    float v = (rand() % 200 - 100) * 0.01;
    data[i] = v;
  }
}
